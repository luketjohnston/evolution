#include "hip/hip_runtime.h"

#include <tclap/CmdLine.h>
#include <bitset>
#include <limits>
#include <ATen/ATen.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "hipblas.h"
#include "debug.h"
#include <time.h> // for srand init
#include "binary.cuh"
#define  torch_device_inttype torch::kInt64
#define  torch_output_inttype torch::kInt32

const bool COMPARE_WITH_CPU{true};
const unsigned int BATCH_SIZE{21}; // TODO should we make threads be max of batch_size and 32?
const unsigned int POPULATION_SIZE{1};
// IN_SIZE must be large enough for a full warp to load integers into memory.
// so it has to be at least 32*32 TODO is this still true?
const unsigned int IN_SIZE{28*28}; // in bits
const unsigned int OUT_SIZE{64}; // in bits

const int DEVICE_INTTYPE_BITS = binary_forward::DEVICE_INTTYPE_BITS;

typedef binary_forward::device_inttype device_inttype;
typedef binary_forward::output_inttype output_inttype;

void printInput(at::Tensor input) {
  const unsigned int population_size = input.sizes()[0];
  const unsigned int batch_size = input.sizes()[1];
  const unsigned int input_size = input.sizes()[2];
  //std::cout << "Sizes:" << input.sizes() << std::endl;
  ///std::cout << "input[p,b,i]:" << input[p,b,i] << std::endl;

  for (int p = 0; p < population_size; p++) {
    for (int b = 0; b < batch_size; b++) {
      for (int i = 0; i < input_size; i++) {
        std::cout << "p" << p << "b" << b << ":" << i << ":" << std::bitset<DEVICE_INTTYPE_BITS>( input.index({p,b,i}).item<device_inttype>()) << std::endl;
      }
    }
  }
}
  
    

void printWeight(at::Tensor weight) {
  const unsigned int population_size = weight.sizes()[1];
  const unsigned int input_size = weight.sizes()[2];
  const unsigned int out_size = weight.sizes()[3];
  for (int p = 0; p < population_size; p++) {
    for (int i = 0; i < input_size; i++) {
      for (int o = 0; o < out_size; o++) {
        for (int t = 0; t < 2; t++) {
          std::cout << "p" << p << "i" << i << "o" << o << "t" << t << ":" << std::bitset<DEVICE_INTTYPE_BITS>( weight.index({t,p,i,o}).item<device_inttype>()) << std::endl;
        }
      }
    }
  }
}

void printOut(at::Tensor out, bool bits) {
  const unsigned int population_size = out.sizes()[0];
  const unsigned int batch_size = out.sizes()[1];
  const unsigned int out_size = out.sizes()[2];
  for (int p = 0; p < population_size; p++) {
    for (int b = 0; b < batch_size; b++) {
      for (int o = 0; o < out_size; o++) {
        if (bits) {
          std::cout << "p" << p << "b" << b << "o" << o << ":" << std::bitset<DEVICE_INTTYPE_BITS>( out.index({p,b,o}).item<device_inttype>()) << std::endl;
        } else {

          std::cout << "p" << p << "b" << b << "o" << o << ":" << out.index({p,b,o}).item<device_inttype>() << std::endl;
        }
      }
    }
  }
}


int main( int argc, char *argv[] )
{
  //argparse::ArgumentParser program("program_name");
  //program.add_argument("-p", "--population").default_value(POPULATION_SIZE);
  //program.add_argument("-i", "--input").default_value(IN_SIZE);
  //program.add_argument("-o", "--output").default_value(OUT_SIZE);
  //program.parse_args(argc, argv);
  //input = program.get<int>("b");


  //unsigned int batch_size = program.get<int>("b");
  //unsigned int population_size = program.get<int>("p");
  //unsigned int in_size = program.get<int>("i");
  //unsigned int out_size = program.get<int>("o");

  TCLAP::CmdLine cmd("TODO descriptor", ' ', "0.1");
  TCLAP::ValueArg<int> batchArg("b","batch_size","Batch size",false,BATCH_SIZE,"int");
  TCLAP::ValueArg<int>   popArg("p","population_size","Population size",false,POPULATION_SIZE,"int");
  TCLAP::ValueArg<int>    inArg("i","in_size","Input size",false,IN_SIZE,"int");
  TCLAP::ValueArg<int>   outArg("o","out_size","Output size",false,OUT_SIZE,"int");

  TCLAP::SwitchArg   vArg("v","verbose","Verbose",false);
  cmd.add(batchArg);
  cmd.add(popArg);
  cmd.add(inArg);
  cmd.add(outArg);
  cmd.add(vArg);
  cmd.parse( argc, argv );




  unsigned int batch_size = batchArg.getValue();
  unsigned int population_size = popArg.getValue();
  unsigned int in_size = inArg.getValue();
  unsigned int out_size = outArg.getValue();
  bool verbose = vArg.getValue();

  //po::options_description desc("Allowed options");
  //desc.add_options()
  //    ("b", po::value<int>(), "Batch size")
  //    ("p", po::value<int>(), "Population size size")
  //    ("i", po::value<int>(), "Input size")
  //    ("o", po::value<int>(), "Output size")
  //;
  //
  //po::variables_map vm;
  //po::store(po::parse_command_line(argc, argv, desc), vm);
  //po::notify(vm);    
  //
  //if (vm.count("b")) {
  //    batch_size = vm["b"].as<int>();
  //} 

  //if (vm.count("p")) {
  //    population_size = vm["p"].as<int>();
  //} 

  unsigned int in_ints = (in_size + DEVICE_INTTYPE_BITS - 1) / DEVICE_INTTYPE_BITS; 


  srand(time(NULL));;
  /* get GPU device number and name */
  int dev;
  hipDeviceProp_t deviceProp;
  checkCUDA( hipGetDevice( &dev ) );
  checkCUDA( hipGetDeviceProperties( &deviceProp, dev ) );
  printf("Using GPU %d: %s\n", dev, deviceProp.name );

  fprintf(stdout, "Input size is %d\n",in_size);
  fprintf(stdout, "Batch size is %d\n",batch_size);
  fprintf(stdout, "Population size is %d\n",population_size);
  fprintf(stdout, "sizeof(int)%lu\n",sizeof(int));
  fprintf(stdout, "sizeof(device_inttype)%lu\n",sizeof(device_inttype));

  auto options =
      torch::TensorOptions()
        .dtype(torch_device_inttype)
        .layout(torch::kStrided)
        .device(torch::kCPU)
        .requires_grad(false);


  const device_inttype maxlong = std::numeric_limits<device_inttype>::max();
  const device_inttype minlong = std::numeric_limits<device_inttype>::min();


  const at::Tensor h_input = torch::randint(minlong,maxlong,{population_size,batch_size,in_ints},options);
  const at::Tensor h_weight = torch::randint(minlong,maxlong,{2,population_size,in_ints,out_size},options);

  const at::Tensor d_input = h_input.to(torch::kCUDA);
  const at::Tensor d_weight = h_weight.to(torch::kCUDA);

  // Note this is different than IN_SIZE / 2 because when IN_SIZE is not divisible
  // by DEVICE_INTTYPE_BITS, we add extend IN_SIZE until it is. 
  const int threshold = in_ints * DEVICE_INTTYPE_BITS / 2;

  // start timers
  hipEvent_t start, stop;
  checkCUDA( hipEventCreate( &start ) );
  checkCUDA( hipEventCreate( &stop ) );
  checkCUDA( hipEventRecord( start, 0 ) );


  /////////
  // GPU //
  /////////

  checkCUDA( hipEventRecord( start, 0 ) );

  at::Tensor d_out_thresh = binary_forward::binary_forward_cuda(
    d_input,
    d_weight,
    threshold,
    verbose);

  at::Tensor d_out_nothresh = binary_forward::binary_forward_cuda(
    d_input,
    d_weight,
    0,
    verbose);

  checkKERNEL();

  // stop timer and print time
  checkCUDA( hipEventRecord( stop, 0 ) );
  checkCUDA( hipEventSynchronize( stop ) );
  float elapsedTime;
  checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
  fprintf(stdout, "Total time GPU is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GBop/s\n", ( ( (double) BATCH_SIZE *
    (double) POPULATION_SIZE * 
    (double) OUT_SIZE * (double) IN_SIZE * 2.0 / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 ))); // TODO check this computation , havent checked

  if (COMPARE_WITH_CPU) {
    // start timer
    checkCUDA( hipEventRecord( start, 0 ) );

  
    // do convolution on cpu
    at::Tensor h_out_thresh = binary_forward::host_helper(h_input, h_weight, threshold, verbose);
    at::Tensor h_out_nothresh = binary_forward::host_helper(h_input, h_weight, 0, verbose);
    //at::Tensor h_out_nothresh = binary_forward::host_helper(h_input, h_weight, threshold);
  
    // stop timers
    checkCUDA( hipEventRecord( stop, 0 ) );
    checkCUDA( hipEventSynchronize( stop ) );
    checkCUDA( hipEventElapsedTime( &elapsedTime, start, stop ) );
  
    // print time taken
    fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
    checkCUDA( hipEventDestroy( start ) );
    checkCUDA( hipEventDestroy( stop ) );
  
    // compare GPU implementation results with CPU results

    device_inttype diff1 = torch::sum(torch::abs(h_out_thresh - d_out_thresh.to(torch::kCPU))).item<device_inttype>();
    device_inttype diff2 = torch::sum(torch::abs(h_out_nothresh - d_out_nothresh.to(torch::kCPU))).item<device_inttype>();

    if (verbose) {
      std::cout << "Input:" << std::endl;
      printInput(h_input);

      //printf("Weight:\n");
      //printWeight(h_weight);
      //printf("Device weight:\n");
      //printWeight(d_weight.to(torch::kCPU));
      std::cout << "Host out:" << std::endl << h_out_thresh << std::endl;
      printOut(h_out_thresh, true);
      std::cout << "Device out:" << std::endl << d_out_thresh.to(torch::kCPU) << std::endl;
      printOut(d_out_thresh.to(torch::kCPU), true);
      std::cout << "NOTHRESH Host out:" << std::endl;
      printOut(h_out_nothresh, false);
      std::cout << "Device NOTHRESH out:" << std::endl;
      printOut(d_out_nothresh.to(torch::kCPU), false);
    }

    
    printf("Threshold error is %ld\n",diff1);
    printf("No threshold error is %ld\n",diff2);

    if( diff1 == 0  && diff2 == 0 ) printf("PASS\n");
    else printf("FAIL\n");


  }

    

  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
      fprintf(stderr, "hipDeviceReset failed!");
      return 1;
  }

  return 0;
}
